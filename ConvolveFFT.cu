#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include "helper.h"

typedef float2 Complex;

__global__ void ComplexMUL(Complex *A, Complex *B, Complex *C, float scale)
{
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
  if (idx < scale){
    hipfftComplex result;
    hipfftComplex mA = A[idx];
    hipfftComplex mB = B[idx];
    result.x = (mA.x * mB.x - mA.y * mB.y)/scale;
    result.y = (mA.x * mB.y + mA.y * mB.x)/scale;
    C[idx] = result;}

}

__global__ void  copyVal(float*out_red,float*out_green,float*out_blue,hipfftComplex *in_red, hipfftComplex *in_green,hipfftComplex *in_blue,int w, int h, int W, int H ){
//__global__ void  copyVal(float*out_red,hipfftComplex *in_red,int w, int h, int W, int H ){

	int ix = threadIdx.x + blockDim.x * blockIdx.x;//xaxis of imagein
    int iy = threadIdx.y + blockDim.y * blockIdx.y;//yaxis of imagein
    int iz = threadIdx.z + blockDim.z * blockIdx.z;	//channels imagein
	
	int currentlocation = iz*w*h + ix + iy * w;

	if (ix < w && iy <h && iz < 1){
	out_red[currentlocation]=in_red[W-w+ix+(H-h+iy)*W].x;
	out_blue[currentlocation]=in_blue[W-w+ix+(H-h+iy)*W].x;
	out_green[currentlocation]=in_green[W-w+ix+(H-h+iy)*W].x;
	}

}

__global__ void pad(float*out_red,float*out_green,float*out_blue,float*in_red, float*in_green, float*in_blue, int w, int h, int W,float* out_kernel, float* in_kernel,int wk,int hk){

//__global__ void pad(float*out_red,float*in_red, int w, int h, int W,float* out_kernel, float* in_kernel,int wk,int hk){



	int ix = threadIdx.x + blockDim.x * blockIdx.x;//xaxis of imagein
    int iy = threadIdx.y + blockDim.y * blockIdx.y;//yaxis of imagein
    int iz = threadIdx.z + blockDim.z * blockIdx.z;	//channels imagein
	
	int currentlocation = iz*w*h + ix + iy * w;
	//printf("Hi\n");
	if (ix < w && iy <h && iz < 1){
			
	
		out_red[ix + iy * W]=in_red[currentlocation];
		out_green[ix + iy * W]=in_green[currentlocation];
		out_blue[ix + iy * W]=in_blue[currentlocation];


	} 

	if(ix < wk && iy <hk && iz < 1)
			out_kernel[ix + iy * W]=in_kernel[iz*wk*hk + ix + iy * wk];
		//TEST	out_kernel[ix + iy * W]=1;
}

__global__ void ConvertComplex(float* in_red,float* in_green,float* in_blue,hipfftComplex *out_red,hipfftComplex *out_green,hipfftComplex *out_blue, int w, int h,float*in_kernel,hipfftComplex *out_kernel){
//__global__ void ConvertComplex(float* in_red,hipfftComplex *out_red, int w, int h,float*in_kernel,hipfftComplex *out_kernel){
	int ix = threadIdx.x + blockDim.x * blockIdx.x;//xaxis of imagein
    int iy = threadIdx.y + blockDim.y * blockIdx.y;//yaxis of imagein
    int iz = threadIdx.z + blockDim.z * blockIdx.z;	//channels imagein
	
	int currentlocation = iz*w*h + ix + iy * w;

	if (ix < w && iy <h && iz < 1){
		out_red[currentlocation].x=in_red[currentlocation];
		out_blue[currentlocation].x=in_blue[currentlocation];
		out_green[currentlocation].x=in_green[currentlocation];
		out_kernel[currentlocation].x=in_kernel[currentlocation];

		out_red[currentlocation].y=0;
		out_blue[currentlocation].y=0;
		out_green[currentlocation].y=0;
		out_kernel[currentlocation].y=0;
	}
}


cv::Mat conv_CUFFT(cv::Mat src, cv::Mat kernel){

	int w=src.cols;
	int h=src.rows;
	int nc=src.channels();
		
	int krows=kernel.rows;
	int kcols=kernel.cols;

	//Split the images to channel
	cv::Mat src_channels[3];
	cv::split(src, src_channels);

	float *R_src_channels = new float[(size_t)w*h];
	float *B_src_channels = new float[(size_t)w*h];
	float *G_src_channels = new float[(size_t)w*h];

	float *kernelArray = new float[(size_t)krows*kcols];
	convert_mat_to_layered (R_src_channels, src);
	convert_mat_to_layered (R_src_channels, src_channels[0]);
	convert_mat_to_layered (B_src_channels,  src_channels[1]);
	convert_mat_to_layered (G_src_channels, src_channels[2]);

	convert_mat_to_layered (kernelArray, kernel);
	////////////////////////////////////////////////////////////////////////////////////
	
	float* pad_red,*pad_blue, *pad_green,*pad_kernel;
 
	int n=(h+2*(krows/2))*(w+2*(kcols/2));
	int nk=krows*kcols;
	hipMalloc(&pad_red,n*sizeof(float));
	hipMemset(pad_red,0, n*sizeof(float));
	hipMalloc(&pad_blue,n*sizeof(float));
	hipMemset(pad_blue,0, n*sizeof(float));
	
	hipMalloc(&pad_green,n*sizeof(float));
	hipMemset(pad_green,0, n*sizeof(float));

	hipMalloc(&pad_kernel,n*sizeof(float));
	hipMemset(pad_kernel,0, n*sizeof(float));

	float* R_src,*G_src, *B_src, *kernel_src;
	hipMalloc(&R_src,h*w*sizeof(float));
	hipMemcpy(R_src, R_src_channels,w*h*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&G_src,h*w*sizeof(float));
	hipMemcpy(G_src, G_src_channels,w*h*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&B_src,h*w*sizeof(float));
	hipMemcpy(B_src, B_src_channels,w*h*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&kernel_src,krows*kcols*sizeof(float));
	hipMemcpy(kernel_src, kernelArray,krows*kcols*sizeof(float), hipMemcpyHostToDevice);

	dim3 Block = dim3(32,32,1);
    dim3 Grid = dim3((w +Block.x -1) / Block.x, (h + Block.y -1) / Block.y, (1+ Block.z -1) / Block.z);
	
	pad<<< Grid,Block >>>(pad_red,pad_green,pad_blue,R_src,G_src,B_src,w,h, w+2*(kcols/2),pad_kernel,kernel_src,kcols,krows);
	
	int W = w+2*(kcols/2);		
	//pad<<< Grid,Block >>>(pad_red,R_src,w,h, W,pad_kernel,kernel_src,kcols,krows);
	


////////////////////////////////////////////////////////////////////////////////
//Convert to Complex
///////////////////////////////////////////////////////////////////////////////


	hipfftComplex* pad_red_complex,*pad_blue_complex, *pad_green_complex,*pad_kernel_complex;
	
	hipMalloc(&pad_red_complex,n*sizeof(Complex));
	hipMalloc(&pad_blue_complex,n*sizeof(hipfftComplex));
	hipMalloc(&pad_green_complex,n*sizeof(hipfftComplex));
	
	hipMalloc(&pad_kernel_complex,n*sizeof(Complex));
ConvertComplex<<<Grid,Block>>>(pad_red,pad_green,pad_blue,pad_red_complex,pad_green_complex,pad_blue_complex, w+2*(kcols/2),h+2*(krows/2),pad_kernel,pad_kernel_complex);

	//ConvertComplex<<<Grid,Block>>>(pad_red,pad_red_complex, w+2*(kcols/2),h+2*(krows/2),pad_kernel,pad_kernel_complex);


// initialize CUFFT library
  	hipfftHandle plan;

  	hipfftPlan1d(&plan, n ,HIPFFT_C2C, 1);

	printf("Transforming signal hipfftExecR2C\n");
    hipfftExecC2C(plan, (hipfftComplex *)pad_red_complex, (hipfftComplex *)pad_red_complex, HIPFFT_FORWARD);
  hipfftExecC2C(plan, (hipfftComplex *)pad_green_complex, (hipfftComplex *)pad_green_complex, HIPFFT_FORWARD);
  	hipfftExecC2C(plan, (hipfftComplex *)pad_blue_complex, (hipfftComplex *)pad_blue_complex, HIPFFT_FORWARD);
    
	hipfftExecC2C(plan, (hipfftComplex *)pad_kernel_complex, (hipfftComplex *)pad_kernel_complex, HIPFFT_FORWARD);


	hipfftComplex *g_RedOut, *g_BlueOut, *g_GreenOut;

	hipMalloc(&g_RedOut, n*sizeof(Complex));
  	hipMalloc(&g_BlueOut, n*sizeof(hipfftComplex));
  	hipMalloc(&g_GreenOut,n*sizeof(hipfftComplex));

	ComplexMUL<<<32,8>>>(pad_kernel_complex, pad_red_complex, g_RedOut, n);
	 ComplexMUL<<<32,8>>>(pad_kernel_complex, pad_green_complex, g_GreenOut, n);
	 ComplexMUL<<<32,8>>>(pad_kernel_complex, pad_blue_complex, g_BlueOut,  n);
 
	hipfftExecC2C(plan, (hipfftComplex *)g_RedOut,(hipfftComplex *) g_RedOut, HIPFFT_BACKWARD);
	hipfftExecC2C(plan, g_GreenOut, g_GreenOut, HIPFFT_BACKWARD);
	hipfftExecC2C(plan, g_BlueOut, g_BlueOut, HIPFFT_BACKWARD);
	
	copyVal<<<Grid, Block>>>(R_src,G_src,B_src,g_RedOut, g_GreenOut, g_BlueOut,w,h, w+2*(kcols/2),h+2*(krows/2));

	copyVal<<<Grid, Block>>>(R_src,g_RedOut, w,h, w+2*(kcols/2),h+2*(krows/2));

 	hipMemcpy(R_src_channels, R_src,w*h*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(G_src_channels, G_src, w*h*sizeof(float), hipMemcpyDeviceToHost);	
	hipMemcpy(B_src_channels, B_src, w*h*sizeof(float), hipMemcpyDeviceToHost);


	convert_layered_to_mat(src_channels[0], R_src_channels);
	convert_layered_to_mat(src_channels[1], B_src_channels);
	convert_layered_to_mat(src_channels[2], G_src_channels);
	
	cv::Mat ImgOut;	
	cv::merge(src_channels,3,ImgOut);


convert_layered_to_mat(src, R_src_channels);
	hipfftDestroy(plan);
 
	return ImgOut;

}

