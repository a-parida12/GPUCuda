#include "library.h"
#include "hipblas.h"

using namespace std;

float MaxFinder(cv::Mat src){
	float maximum;
	int w=src.cols;
	int h=src.rows;
	int nc=src.channels();
	
	float *srcArray = new float[(size_t)w*h*nc];
	float *g_srcArray;
	hipMalloc( &g_srcArray, w*h*nc * sizeof(float) );
	convert_mat_to_layered ( srcArray,src);
	hipMemcpy( g_srcArray, srcArray, w*h*nc * sizeof(float), hipMemcpyHostToDevice);

	int max_id=hipblasIsamax(h*w*nc,g_srcArray,1);
	
	maximum=srcArray[max_id-1];
	hipFree(g_srcArray);
	delete[] srcArray;
	return maximum;

}
