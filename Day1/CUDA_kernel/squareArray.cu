// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__device__ float square(float a) {return a*a;}

__global__ void g_square(float*a, int n)
	{
		int ind = threadIdx.x + blockDim.x * blockIdx.x;
		if(ind<n) a[ind]=square(a[ind]);
	}

int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 10;
    float *a = new float[n];
    for(int i=0; i<n; i++) a[i] = i;

    // CPU computation
    for(int i=0; i<n; i++)
    {
        float val = a[i];
        val = val*val;
        a[i] = val;
    }

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;
    
    // GPU computation
    // reinit data
    for(int i=0; i<n; i++) a[i] = i;

	// ###
    // ### TODO: Implement the "square array" operation on the GPU and store the result in "a"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"


	dim3 block = dim3(10,1,1);
	dim3 grid = dim3((n+block.x-1)/block.x,1,1);	

	float *d_a;
	hipMalloc(&d_a,n*sizeof(float));
	CUDA_CHECK;
	
	
	hipMemcpy( d_a, a, n * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
	
	g_square<<<grid,block>>>(d_a,n);

	hipMemcpy( a, d_a, n * sizeof(float), hipMemcpyDeviceToHost );CUDA_CHECK;
	hipFree(d_a);CUDA_CHECK;

    
    // print result
    cout << "GPU:" << endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
}



