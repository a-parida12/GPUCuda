// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;



// cuda error checking
#define CUDA_CHECK cuda_check(__FILE__,__LINE__)
void cuda_check(string file, int line)
{
    hipError_t e = hipGetLastError();
    if (e != hipSuccess)
    {
        cout << endl << file << ", line " << line << ": " << hipGetErrorString(e) << " (" << e << ")" << endl;
        exit(1);
    }
}

__device__ float add(float a, float b)
{
  return a + b;
}

__global__ void vecAdd(float *a,float *b,float *c,int n)
{ 	

	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if(idx<n)
		c[idx]=add(a[idx],b[idx]);
}		



int main(int argc, char **argv)
{
    // alloc and init input arrays on host (CPU)
    int n = 20;
    float *a = new float[n];
    float *b = new float[n];
    float *c = new float[n];
    for(int i=0; i<n; i++)
    {
        a[i] = i;
        b[i] = (i%5)+1;
        c[i] = 0;
    }

    // CPU computation
    for(int i=0; i<n; i++) c[i] = a[i] + b[i];

    // print result
    cout << "CPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;
    // init c
    for(int i=0; i<n; i++) c[i] = 0;
    
	

    // GPU computation
    // ###
    // ### TODO: Implement the array addition on the GPU, store the result in "c"
    // ###
    // ### Notes:
    // ### 1. Remember to free all GPU arrays after the computation
    // ### 2. Always use the macro CUDA_CHECK after each CUDA call, e.g. "cudaMalloc(...); CUDA_CHECK;"
    // ###    For convenience this macro is defined directly in this file, later we will only include "helper.h"
    
	
	float *d_a,*d_b,*d_c;
	hipMalloc(&d_a,n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&d_b,n*sizeof(float));
	CUDA_CHECK;
	hipMalloc(&d_c,n*sizeof(float));
	CUDA_CHECK;
	
	hipMemcpy( d_a, a, n * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
	hipMemcpy( d_b, b, n * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
	hipMemcpy( d_c, c, n * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
	
	dim3 block = dim3(128,1,1);
	dim3 grid = dim3((n+block.x-1)/ block.x,1,1);
	
	vecAdd<<<grid,block>>>(d_a,d_b,d_c,n);

	hipMemcpy( c, d_c, n * sizeof(float), hipMemcpyDeviceToHost );CUDA_CHECK;
	

	// print result
    cout << "GPU:"<<endl;
    for(int i=0; i<n; i++) cout << i << ": " << a[i] << " + " << b[i] << " = " << c[i] << endl;
    cout << endl;

    // free CPU arrays
    delete[] a;
    delete[] b;
    delete[] c;

	//free GPU arrays
	
	hipFree(d_a);CUDA_CHECK;
	hipFree(d_b);CUDA_CHECK;
	hipFree(d_c);CUDA_CHECK;
	
}



