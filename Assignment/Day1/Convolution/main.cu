#include "hip/hip_runtime.h"

// ###
// ###
// ### Practical Course: GPU Programming in Computer Vision
// ###
// ###
// ### Technical University Munich, Computer Vision Group
// ### Summer Semester 2017, September 11 - October 9
// ###

#include "helper.h"
#include <iostream>
#include <stdio.h>
using namespace std;

// uncomment to use the camera
//#define CAMERA

__global__ void convoluteGPU (float *in, float *out, int w, int h, int nc, float *kernel, int kernelRadius)
{
	int ix = threadIdx.x + blockDim.x * blockIdx.x;//xaxis of imagein
    int iy = threadIdx.y + blockDim.y * blockIdx.y;//yaxis of imagein
    int iz = threadIdx.z + blockDim.z * blockIdx.z;	//channels imagein
	//printf("thread id check\n");
	int kernelWidth = 2 * kernelRadius + 1;
    
    int currentlocation = iz*w*h + ix + iy * w;
    out[currentlocation]=0;
	//printf("current location check\n");
	if (ix < w && iy <h && iz < nc){ 

		for (int x=0; x<kernelWidth; x++) {
			for(int y=0; y<kernelWidth; y++) {

				int cx = max(min(w-1, ix + x - kernelRadius), 0);
				int cy = max(min(h-1, iy + y - kernelRadius), 0);

				out[currentlocation] += kernel[x+y*kernelWidth] * in[iz*w*h+cx+cy*w];
			}
		}
	}
}


int main(int argc, char **argv)
{
    // Before the GPU can process your kernels, a so called "CUDA context" must be initialized
    // This happens on the very first call to a CUDA function, and takes some time (around half a second)
    // We will do it right here, so that the run time measurements are accurate
    hipDeviceSynchronize();  CUDA_CHECK;




    // Reading command line parameters:
    // getParam("param", var, argc, argv) looks whether "-param xyz" is specified, and if so stores the value "xyz" in "var"
    // If "-param" is not specified, the value of "var" remains unchanged
    //
    // return value: getParam("param", ...) returns true if "-param" is specified, and false otherwise

#ifdef CAMERA
#else
    // input image
    string image = "";
    bool ret = getParam("i", image, argc, argv);
    if (!ret) cerr << "ERROR: no image specified" << endl;
    if (argc <= 1) { cout << "Usage: " << argv[0] << " -i <image> [-repeats <repeats>] [-gray]" << endl; return 1; }
#endif
    
    // number of computation repetitions to get a better run time measurement
    int repeats = 1;
    getParam("repeats", repeats, argc, argv);
    cout << "repeats: " << repeats << endl;
    
    // load the input image as grayscale if "-gray" is specifed
    bool gray = false;
    getParam("gray", gray, argc, argv);
    cout << "gray: " << gray << endl;

    // ### Define your own parameters here as needed    

    // Init camera / Load input image
#ifdef CAMERA

    // Init camera
  	cv::VideoCapture camera(0);
  	if(!camera.isOpened()) { cerr << "ERROR: Could not open camera" << endl; return 1; }
    int camW = 640;
    int camH = 480;
  	camera.set(CV_CAP_PROP_FRAME_WIDTH,camW);
  	camera.set(CV_CAP_PROP_FRAME_HEIGHT,camH);
    // read in first frame to get the dimensions
    cv::Mat mIn;
    camera >> mIn;
    
#else
    
    // Load the input image using opencv (load as grayscale if "gray==true", otherwise as is (may be color or grayscale))
    cv::Mat mIn = cv::imread(image.c_str(), (gray? CV_LOAD_IMAGE_GRAYSCALE : -1));
    // check
    if (mIn.data == NULL) { cerr << "ERROR: Could not load image " << image << endl; return 1; }
    
#endif

    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
    // get image dimensions
    int w = mIn.cols;         // width
    int h = mIn.rows;         // height
    int nc = mIn.channels();  // number of channels
    cout << "image: " << w << " x " << h << endl;




    // Set the output image format
    // ###
    // ###
    // ### TODO: Change the output image format as needed
    // ###
    // ###
    cv::Mat mOut(h,w,mIn.type());  // mOut will have the same number of channels as the input image, nc layers
    //cv::Mat mOut(h,w,CV_32FC3);    // mOut will be a color image, 3 layers
    //cv::Mat mOut_gray(h,w,CV_32FC1);    // mOut will be a grayscale image, 1 layer
    // ### Define your own output images here as needed




    // Allocate arrays
    // input/output image width: w
    // input/output image height: h
    // input image number of channels: nc
    // output image number of channels: mOut.channels(), as defined above (nc, 3, or 1)

    // allocate raw input image array
    float *imgIn = new float[(size_t)w*h*nc];

    // allocate raw output array (the computation result will be stored in this array, then later converted to mOut for displaying)
	float *imgOut = new float[(size_t)w*h*mOut.channels()];
	float *imgOutConvolutedCPU = new float[(size_t)w*h*mOut.channels()];
	
	    // For camera mode: Make a loop to read in camera frames
#ifdef CAMERA
    // Read a camera image frame every 30 milliseconds:
    // cv::waitKey(30) waits 30 milliseconds for a keyboard input,
    // returns a value <0 if no key is pressed during this time, returns immediately with a value >=0 if a key is pressed
    while (cv::waitKey(30) < 0)
    {
    // Get camera image
    camera >> mIn;
    // convert to float representation (opencv loads image values as single bytes by default)
    mIn.convertTo(mIn,CV_32F);
    // convert range of each channel to [0,1] (opencv default is [0,255])
    mIn /= 255.f;
#endif

    // Init raw input image array
    // opencv images are interleaved: rgb rgb rgb...  (actually bgr bgr bgr...)
    // But for CUDA it's better to work with layered images: rrr... ggg... bbb...
    // So we will convert as necessary, using interleaved "cv::Mat" for loading/saving/displaying, and layered "float*" for CUDA computations
    convert_mat_to_layered (imgIn, mIn);

    
    // ###
    // ###
    // ### TODO: Main computation
    // ###
    // ###

	// create kernel	
	
	
    float sigma=1.0;


	int radius_kernel=ceil(3*sigma);
	int width_kernel=2*radius_kernel+1;
	float sum=0.0;
	float sigmasquare_x2 = 2.0 * sigma * sigma;
	float gKernel[width_kernel*width_kernel];
	float mKernel=0.0;

	for (int x=0;x<width_kernel;x++)
	{
	 	for(int y=0;y<width_kernel;y++)
		{
			int a= x-radius_kernel;
			int b= y-radius_kernel;
            gKernel[x+y*width_kernel] = expf(-(a*a+b*b)/sigmasquare_x2)/(M_PI * sigmasquare_x2);
            sum += gKernel[x+y*width_kernel];
			
			if(gKernel[x+y*width_kernel] > mKernel){
				mKernel = gKernel[x+y*width_kernel];
			}
			
        }													
	
	}
	float copy_gKernel[width_kernel*width_kernel];
    for(int i = 0; i < width_kernel; ++i)
    {
        for (int j = 0; j < width_kernel; ++j)
            {
			gKernel[i+j*width_kernel]/=sum;
			//cout<<gKernel[i+j*width_kernel]<<"\t";
			copy_gKernel[i+j*width_kernel]=gKernel[i+j*width_kernel]/gKernel[width_kernel/2+(width_kernel/2)*width_kernel];
        }
		//cout<<endl;
    }

	 cv::Mat mOutKernel(width_kernel,width_kernel,CV_32FC1);
    
    convert_layered_to_mat(mOutKernel, copy_gKernel);
    showImage("Gaussian Kernel", mOutKernel, 250, 100);
	

// apply convolution with clamping
	Timer timer; timer.start();
	for(int c=0; c<nc; c++) {
		

		for (int ix=0; ix<w; ix++) {
		    for(int iy=0; iy<h; iy++) {

				int currentlocation = h*w*c +ix+iy*w;

		    	for (int x=0; x<width_kernel; x++) {
					for(int y=0; y<width_kernel; y++) {

						//clamping strategy
					    int cx = max(min(w-1, ix + x - radius_kernel), 0);
					    int cy = max(min(h-1, iy + y - radius_kernel), 0);

						imgOutConvolutedCPU[currentlocation] += gKernel[x+y*width_kernel] * imgIn[h*w*c+cx+cy*w];
	//cout<<imgOutConvolutedCPU[currentlocation]<<endl;
					}
				}
			}
		}
	}
	
	 timer.end();  float t = timer.get();  // elapsed time in seconds
    cout << "time on CPU: " << t*1000 << " ms" << endl;

    convert_layered_to_mat(mOut, imgOutConvolutedCPU);
    showImage("Output Convoluted CPU", mOut, 150, 100);
	
   //--Init for Cuda kernel call

	float *imgOutConvolutedGPU = new float[(size_t)w*h*nc];
    
    float *g_imgIn;
    float *g_imgOut;
    float *g_gKernel;
    
    hipMalloc( &g_imgIn, w*h*nc * sizeof(float) );CUDA_CHECK;
    hipMalloc( &g_imgOut, w*h*nc * sizeof(float) );CUDA_CHECK;
    hipMalloc( &g_gKernel, width_kernel * width_kernel * sizeof(float) );CUDA_CHECK;
    
    hipMemcpy( g_imgIn, imgIn, w*h*nc * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK; 
    hipMemcpy( g_gKernel, gKernel, width_kernel * width_kernel * sizeof(float), hipMemcpyHostToDevice );CUDA_CHECK;
    
    dim3 Block = dim3(32,32,1);
    dim3 Grid = dim3((w +Block.x -1) / Block.x, (h + Block.y -1) / Block.y, (nc+ Block.z -1) / Block.z);
    				
	//call cuda kernel for convolution

	Timer timer1; timer1.start();
	convoluteGPU <<<Grid,Block>>> (g_imgIn, g_imgOut, w, h, nc, g_gKernel, radius_kernel);CUDA_CHECK;
    timer1.end();  t = timer1.get();  // elapsed time in seconds
    cout << "time on GPU: " << t*1000 << " ms" << endl;

	//copy output gpu->cpu
    hipMemcpy(imgOutConvolutedGPU,g_imgOut, nc*h*w * sizeof(float), hipMemcpyDeviceToHost );
    CUDA_CHECK;
    
    //free gpu allocation
    hipFree(g_imgOut);
    CUDA_CHECK;
    hipFree(g_imgIn);
    CUDA_CHECK;
    hipFree(g_gKernel);
    CUDA_CHECK;

	convert_layered_to_mat(mOut, imgOutConvolutedGPU);
    showImage("Output Convoluted GPU", mOut, 200, 100);

	convert_layered_to_mat(mOut, imgIn);
    showImage("Input Image", mOut, 250, 100);

#ifdef CAMERA
    // end of camera loop
    }
#else
    // wait for key inputs
    cv::waitKey(0);
#endif




    // save input and result
    //cv::imwrite("image_input.png",mIn*255.f);  // "imwrite" assumes channel range [0,255]
    //cv::imwrite("image_result.png",mOut*255.f);

    // free allocated arrays
    delete[] imgIn;
    delete[] imgOut;

    // close all opencv windows
    cvDestroyAllWindows();
    return 0;
}



